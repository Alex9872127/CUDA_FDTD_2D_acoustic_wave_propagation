/*
Hello world of wave propagation in CUDA. FDTD acoustic wave propagation in homogeneous medium. Second order in space and time 
*/

#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include "string.h"
/*
Add this to c_cpp_properties.json if linting isn't working for cuda libraries
"includePath": [
                "/usr/local/cuda-9.0/targets/x86_64-linux/include",
                "${workspaceFolder}/**"
            ],
*/          
#include "hip/hip_runtime.h"



// Check error codes for CUDA functions
#define CHECK(call)                                                            \
{                                                                              \
    hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

#define PI      3.14159265359
#define PAD     4
#define a0     -3.0124472f
#define a1      1.7383092f
#define a2     -0.2796695f
#define a3      0.0547837f
#define a4     -0.0073118f

// Allocate the constant device memory
__constant__ float c_coef[5];       /* coefficients for 8th order fd */
__constant__ int c_isrc;            /* source location, ox */
__constant__ int c_jsrc;            /* source location, oz */
__constant__ int c_nx;              /* x dim */
__constant__ int c_ny;              /* y dim */

// Add source wavelet
__global__ void kernel_add_wavelet(float *d_u1, float *d_wavelet)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * c_nx + ix;

    if (ix == c_isrc) {
        printf("!!!%i",ix);
    }
    
    printf("GPU kernel add wavelet %i \n", idx);
}

// FD kernel
__global__ void kernel_2dfd(float *d_u1, float *d_u2, float *d_vp)
{

}

int main( int argc, char *argv[])
{
    // Print out name of the main GPU
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, 0));
    printf("%d: %s\n", 0, deviceProp.name);
    CHECK(hipSetDevice(0));

    // Model dimensions
    int nx    = 1024;                       /* x dim */
    int ny    = 1024;                       /* z dim */

    // Add padding for derivatives
    nx += 2 * PAD;
    ny += 2 * PAD;

    size_t nxy = nx * ny;   
    size_t nbytes = nxy * sizeof(float);    /* bytes to store nx * ny */
    
    float dx = 1;                           /* m */
    float dy = dx;
    
    // Allocate memory for velocity model
    float _vp = 3300;                       /* m/s, p-wave velocity */
    float *h_vp;
    h_vp = (float *)malloc(nbytes);
    memset(h_vp, _vp, nbytes);              /* initiate h_vp with _vp */

    // Time stepping
    float t_total = 0.05;                   /* sec, total time of wave propagation */
    float dt = 0.7 * fmin(dx, dy) / _vp;    /* sec, time step assuming constant vp */
    int nt = round(t_total / dt);         /* number of time steps */

    // Source
    float f0 = 10.0;                        /* Hz, source dominant frequency */
    float t0 = 1.2 / f0;                    /* source padding to move wavelet from left of zero */

    float *h_wavelet, *h_time;
    h_time = (float *) malloc(nt * sizeof(float));
    h_wavelet = (float *) malloc(nt * sizeof(float));

    // Fill source waveform vecror
    float a = PI * PI * f0 * f0;            /* const for wavelet */
    for(size_t it = 0; it < nt; it++)
    {
        h_time[it] = it * dt;
        h_wavelet[it] = 1e10 * (1.0 - 2.0*a*pow(h_time[it] - t0, 2))*exp(-a*pow(h_time[it] - t0, 2));
        h_wavelet[it] *= dt * dt / (dx * dy);
    }

    // Allocate memory on device
    float *d_u1, *d_u2, *d_vp, *d_wavelet;
    CHECK(hipMalloc((void **) &d_u1, nbytes))          /* wavefield at t-1 */
    CHECK(hipMalloc((void **) &d_u2, nbytes))          /* wavefield at t-2 */
    CHECK(hipMalloc((void **) &d_vp, nbytes))          /* velocity model */
    CHECK(hipMalloc((void **) &d_wavelet, nbytes));    /* source term for each time step */
    
    // Transfer data to device
    CHECK(hipMemset(d_u1, 0, nbytes))
    CHECK(hipMemset(d_u2, 0, nbytes))
    CHECK(hipMemcpy(d_vp, h_vp, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_wavelet, h_wavelet, nbytes, hipMemcpyHostToDevice));

    float coef[] = {a0, a1, a2, a3, a4};
    int isrc = round((float) nx / 2);                 /* source location, ox */
    int jsrc = round((float) ny / 2);                 /* source location, oz */

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_coef), coef, 5 * sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_isrc), &isrc, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_jsrc), &jsrc, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nx), &nx, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_ny), &ny, sizeof(int)));
    
    printf("%i\tnt\n",nt);
    for(int istep = 0; istep < nt; istep++)
    {
    //    kernel_2dfd<<<grid, block>>>(d_u1, d_u2)
        kernel_add_wavelet<<<32,1>>>(d_u1, d_wavelet);
    }
    
    CHECK(hipFree(d_u1));
    CHECK(hipFree(d_u2));
    CHECK(hipFree(d_vp));
    CHECK(hipFree(d_wavelet));

    CHECK(hipDeviceReset());



    return 0;
}
